#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform_scan.h>
#include <thrust/partition.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <iterator>
#include <sstream>
#include <chrono>
#include "common.cuh"
#include <stack>

using namespace std;

bool is_opening_bracket(const char& c)
{
  if(c=='{' || c=='[') return true;
  return false;
}

bool is_closing_bracket(const char& c)
{
  if(c=='}' || c==']') return true;
  return false;
}

bool is_bracket_coresponding(const char& c, const char& c_stack)
{
  if(c=='}') 
  {
    if(c_stack=='{') return true;
  }
  else if(c==']')
  {
    if(c_stack=='[') return true;
  }
  return false;
}

__host__
bool h_is_balanced_parentheses(const string& s)
{
    stack<char> bracket_stack;
    int depth = 0;

    for (auto it = s.cbegin() ; it != s.cend(); ++it) 
    {
      if(is_opening_bracket(*it))
      {
        bracket_stack.push(*it);
        depth++;
      }
      else if(is_closing_bracket(*it))
      {
        if(is_bracket_coresponding(*it,bracket_stack.top()))
        {
          depth--;
          if(depth<0) return false;
          bracket_stack.pop();
        }
        else return false;
      }
    }
    if(depth==0) return true;
    return false;
}

__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid; 
}

struct data
{
  thrust::device_vector<char> chars;
  thrust::device_vector<short> positions;
  thrust::device_vector<short> levels;
};

int main(int argc, char **argv)
{
  cout << "CUDA JSON Validator" << endl << endl;
  cout << "Warming up" << endl << endl;
  warm_up_gpu<<<1024, 1024>>>();


  string result = "";

  auto startReadingFile = chrono::steady_clock::now();

  // Reading file
  const string& json_file = argc == 2 ? argv[1] : "samples/senators_short.json";
  ifstream ifile(json_file);

  ostringstream ss;
  ss << ifile.rdbuf();
  const string& s = ss.str();

  elapsedTime(startReadingFile, "Reading file");

  auto startCpu = chrono::steady_clock::now();
  //cpu
  string result_cpu = h_is_balanced_parentheses(s) ? "correct" : "wrong";
  cout << "\nCpu algorithm claims that file is " << result_cpu <<endl;
  elapsedTime(startCpu, "Cpu alogithm");

  auto startCopying = chrono::steady_clock::now();

  thrust::host_vector<char> H_file(s.begin(), s.end());
  thrust::device_vector<char> D_file(H_file.size());

  size_t file_size = H_file.size();

  // Copy file to device memory
  thrust::copy(H_file.begin(), H_file.end(), D_file.begin());

  elapsedTime(startCopying, "Copying file");
  auto startCalculations = chrono::steady_clock::now();

  data dd;

  dd.chars = thrust::device_vector<char>(D_file.size());
  dd.positions = thrust::device_vector<short>(D_file.size());

  auto char_and_pos = thrust::make_zip_iterator(thrust::make_tuple(D_file.begin(), thrust::make_counting_iterator(0)));
  auto d_char_and_pos = thrust::make_zip_iterator(thrust::make_tuple(dd.chars.begin(), dd.positions.begin()));

  auto last_brace_it = thrust::copy_if(char_and_pos, char_and_pos + s.length(), d_char_and_pos, is_brace_or_bracket());
  
  auto chars_count = last_brace_it - d_char_and_pos;
  dd.chars.resize(chars_count);
  dd.positions.resize(chars_count);

  dd.levels = thrust::device_vector<short>(chars_count);

  thrust::transform_inclusive_scan(dd.chars.begin(), dd.chars.end(), dd.levels.begin(), braces_to_numbers2(), thrust::plus<short>());
  thrust::transform_if(dd.levels.begin(), dd.levels.end(), dd.chars.begin(), dd.levels.begin(), increment2(), is_closing_brace2());

  char last_brace_level = dd.levels[dd.levels.size() - 1];

  if(result == "" && last_brace_level != 1){
    stringstream tmp;
    tmp << "Braces or brackets in this JSON are incorrect. Last brace has level " << (int)last_brace_level << ", but should have level 1";
    result = tmp.str();
  }


  auto adjacent_chars = thrust::make_zip_iterator(thrust::make_tuple(dd.chars.begin(), dd.chars.begin() + 1));
  bool are_chars_correct = thrust::all_of(adjacent_chars, adjacent_chars + dd.chars.size() - 1, opening_and_closing_chars_are_corresponding2());

  if(!are_chars_correct){
    result = "Found sequence [} or {]";
  }

  auto chars_and_levels = thrust::make_zip_iterator(thrust::make_tuple(dd.chars.begin(), dd.levels.begin()));
  auto chars_and_levels_end = thrust::make_zip_iterator(thrust::make_tuple(dd.chars.end(), dd.levels.end()));
  thrust::stable_partition(chars_and_levels, chars_and_levels_end, is_brace2());
  auto adjacent_brackets = thrust::make_zip_iterator(thrust::make_tuple(chars_and_levels, chars_and_levels + 1));
  bool are_brackets_correct = thrust::all_of(adjacent_brackets, adjacent_brackets + dd.chars.size(), opening_and_closing_chars_have_the_same_level2());

  if(!are_brackets_correct){
    result = "Something between some brackets is incorrect";
  }

  elapsedTime(startCalculations, "Calculations");

  /*
  for(int i = 0; i < D_json_chars.size(); i++){
    json_char c = (json_char)D_json_chars[i];
    cout << c.position << " - " << c._char << " - " << c.level << endl;
  }
  */


 

  if(result != ""){
    cout << "JSON is incorrect:\n\t" << result << endl;
  } else {
    cout << "JSON is correct" << endl;
  }
  
  return 0;
}
